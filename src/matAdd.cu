
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;

__global__
void simpleMatAddKernel(float *A, float *B, float *C, int side){
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int idx = y * side + x;
    C[idx] = A[idx] + B[idx]; 
}

void matAdd(float *h_A, float *h_B, float *h_C, int side){
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    int matrix_size = side * side * sizeof(float);
    hipMalloc((void **)&d_A, matrix_size);
    hipMalloc((void **)&d_B, matrix_size);
    hipMalloc((void **)&d_C, matrix_size);

    // Copy input data to device
    hipMemcpy(d_A, h_A, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrix_size, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((side + blockDim.x - 1 )/blockDim.x, (side + blockDim.y - 1 )/blockDim.y, 1);
    // TODO: Measure performance
    // TODO: Include one-thread-per-row approach
    // TODO: Include one-thread-per-col approach
    simpleMatAddKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, side);

    // Copy output to host
    hipMemcpy(h_C, d_C, matrix_size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

int main(int argc, char **argv){
    // Define input vectors
    const int SIDE=100;
    vector<float> h_A(SIDE * SIDE, 10.0f);
    vector<float> h_B(SIDE * SIDE, 20.0f);
    vector<float> h_C(SIDE * SIDE, 0.0f);

    // Initialise values for A and B
    for (size_t i=0; i < (SIDE * SIDE); i++){
        // TODO: Generate random numbers here
    }

    // Launch matrix addition
    matAdd(h_A.data(), h_B.data(), h_C.data(), SIDE);

    cout << h_C[0] << endl;

    return 0;
}