#include <chrono>
#include <iostream>
#include <vector>
#include "benchmark_common.cuh"

using namespace std;

void benchmarkSync(float *h_A, float *h_B, float *h_C, int m, int k, int n){
    // Allocate GPU memory
    float *d_A;
    float *d_B;
    float *d_C;
    int sizeA = m * k * sizeof(float);
    int sizeB = k * n * sizeof(float);
    int sizeC = m * n * sizeof(float);
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_A, sizeA));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_B, sizeB));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_C, sizeC));

    // Copy input data to device
    CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice));

    // Start measuring time
    auto start = chrono::steady_clock::now();

    // Launch kernel
    dim3 blockDim(32, 32, 1);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y, 1); 
    benchmark::matMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, k, n);

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Finish measuring time
    auto end = chrono::steady_clock::now();
    auto duration = chrono::duration_cast<chrono::microseconds>(end - start);
    cout << "Kernel took: " << duration.count() / 1000.f << "ms" << endl;

    // Check for sync errors in the kernel launch
    CHECK_LAST_CUDA_ERROR();

    // Copy output to host
    CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost));

    // Free memory
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_C));
}

int main(int argc, char **argv){
    if (argc < 2){
        cerr << "Usage: " << argv[0] << " <MATRIX_SIZE>" << endl;
        return 1;
    }

    int MATRIX_SIZE;
    try {
        MATRIX_SIZE = stoi(argv[1]);
        if (MATRIX_SIZE <= 0) throw invalid_argument("Matrix size must be positive.");
    }
    catch (const exception &e) {
        cerr << "Error: Invalid matrix size: " << e.what() << endl;
        return 1;
    }
    
    // Initialise random matrices
    vector<float> A(MATRIX_SIZE * MATRIX_SIZE);
    vector<float> B(MATRIX_SIZE * MATRIX_SIZE);
    vector<float> C(MATRIX_SIZE * MATRIX_SIZE);
    benchmark::randomizeVector(A);
    benchmark::randomizeVector(B);

    // Run the benchmark
    benchmarkSync(A.data(), B.data(), C.data(), MATRIX_SIZE, MATRIX_SIZE, MATRIX_SIZE);

    return 0;
}